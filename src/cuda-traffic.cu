#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-traffic.c - Biham-Middleton-Levine traffic model
 *
 * Luka Micheletti, Matricola 723450
 * Progetto HPC 2016/2017
 *
 * ---------------------------------------------------------------------------
 *
 * This program implements the Biham-Middleton-Levine traffic model
 * The BML traffic model is a simple three-state 2D cellular automaton
 * over a toroidal square lattice space. Initially, each cell is
 * either empty, or contains a left-to-right (LR) or top-to-bottom
 * (TB) moving vehicle. The model evolves at discrete time steps. Each
 * step is logically divided into two phases: in the first phase only
 * LR vehicles move, provided that the destination cell is empty; in
 * the second phase, only TB vehicles move, again provided that the
 * destination cell is empty.
 *
 * Compile with:
 * nvcc -Wno-deprecated-gpu-targets cuda-traffic.cu -o cuda-traffic
 *
 * Run with:
 * ./cuda-traffic [nsteps [rho [N]]]
 * 
 * where nsteps is the number of simulation steps to execute, rho is
 * the density of vehicles (probability that a cell is occupied by a
 * vehicle), and N is the grid size.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>

#define BUFLEN 256
#define BLKSIZE 16

/* Translates bidimensional indexes to a monodimensional one. 
 * |i| is the column index.
 * |j| is the row index.
 * |n| is the number of columns (length of the rows). */
#define IDX(i, j, n) ((i) * (n) + (j))

typedef unsigned char cell_t;

/* Possible values stored in a grid cell */
enum {
    EMPTY = 0,  /* empty cell            */
    LR,         /* left-to-right vehicle */
    TB          /* top-to-bottom vehicle */
};

/*|grid| points to a (n + 2) * (n + 2) block of bytes; this function copies
  the bottom and top n elements to the opposite ghost cell layer (see figure
  below). 
 
   0 1              n n+1
   | |              | |
   v v              v v
  +-+----------------+-+
  |Y|YYYYYYYYYYYYYYYY|Y| <- 0
  +-+----------------+-+
  |X|XXXXXXXXXXXXXXXX|X| <- 1
  |\|                |\|
  |\|                |\|
  |\|                |\|
  |\|                |\|
  |Y|YYYYYYYYYYYYYYYY|Y| <- n
  +-+----------------+-+
  |X|XXXXXXXXXXXXXXXX|X| <- n+1
  +-+----------------+-+
 */
__global__ void copy_top_bottom(cell_t *grid, int n) {
    const int j = 1 + threadIdx.x + blockIdx.x * blockDim.x;
    if (j < n + 1) {
        grid[IDX(n + 1, j, n + 2)] = grid[IDX(1, j, n + 2)];
        grid[IDX(0, j, n + 2)] = grid[IDX(n, j, n + 2)];
    }    
}

/*|grid| points to a (n + 2) * (n + 2) block of bytes; this function copies
  the left and right (n + 2) elements to the opposite ghost cell layer (see figure
  below).
 
   0 1              n n+1
   | |              | |
   v v              v v
  +-+----------------+-+
  |Y|X\\\\\\\\\\\\\\Y|X| <- 0
  +-+----------------+-+
  |Y|X              Y|X| <- 1
  |Y|X              Y|X|
  |Y|X              Y|X|
  |Y|X              Y|X|
  |Y|X              Y|X|
  |Y|X              Y|X| <- n
  +-+----------------+-+
  |Y|X\\\\\\\\\\\\\\Y|X| <- n+1
  +-+----------------+-+
 */
__global__ void copy_left_right(cell_t *grid, int n) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n + 2) {
        grid[IDX(i, n + 1, n + 2)] = grid[IDX(i, 1, n + 2)];
        grid[IDX(i, 0, n + 2)] = grid[IDX(i, n, n + 2)];
    }
}

/* Move all left-to-right vehicles that are not blocked. */
__global__ void horizontal_step(cell_t *cur, cell_t *next, int n) {
    const int x = 1 + threadIdx.y + blockIdx.y * blockDim.y;
    const int y = 1 + threadIdx.x + blockIdx.x * blockDim.x;

    if (x < n + 1 && y < n + 1) {
        if (cur[IDX(x, y - 1, n + 2)] == LR && cur[IDX(x, y, n + 2)] == EMPTY) {
            next[IDX(x, y, n + 2)] = LR;
        } else if (cur[IDX(x, y, n + 2)] == LR && cur[IDX(x, y + 1, n + 2)] == EMPTY) {
            next[IDX(x, y, n + 2)] = EMPTY;
        } else {
            next[IDX(x, y, n + 2)] = cur[IDX(x, y, n + 2)];
        }
	}
}

/* Move all top-to-bottom vehicles that are not blocked. */
__global__ void vertical_step(cell_t *cur, cell_t *next, int n) {
    const int x = 1 + threadIdx.y + blockIdx.y * blockDim.y;
    const int y = 1 + threadIdx.x + blockIdx.x * blockDim.x;

    if (x < n + 1 && y < n + 1) {
        if (cur[IDX(x - 1, y, n + 2)] == TB && cur[IDX(x, y, n + 2)] == EMPTY) {
            next[IDX(x, y, n + 2)] = TB;
        } else if (cur[IDX(x, y, n + 2)] == TB && cur[IDX(x + 1, y, n + 2)] == EMPTY) {
            next[IDX(x, y, n + 2)] = EMPTY;
        } else {
            next[IDX(x, y, n + 2)] = cur[IDX(x, y, n + 2)];
        }
    }
}

/* Initialize |grid| with vehicles with density |rho|. |rho| must be
   in the range [0, 1] (rho = 0 means no vehicle, rho = 1 means that
   every cell is occupied by a vehicle). The direction is chosen with
   equal probability. */
void setup(cell_t *grid, int n, float rho) {
    int i;
    int j;

    for (i = 1; i < n - 1; i++) {
        for (j = 1; j < n - 1; j++) {
            if (((float) rand() / (float) (RAND_MAX)) < rho) {
                if (rand() % 100 < 50) {
                    grid[IDX(i, j, n)] = LR;
                } else {
                    grid[IDX(i, j, n)] = TB;
                }
            } else {
                grid[IDX(i, j, n)] = EMPTY;
            }
        }
    }
}

/* Dump |grid| as a PPM (Portable PixMap) image written to file
   |filename|. LR vehicles are shown as blue pixels, while TB vehicles
   are shown in red. Empty cells are white. */
void dump(const cell_t *grid, int n, const char *filename) {
    int i;
    int j;
    FILE *out = fopen( filename, "w" );
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n - 2, n - 2);
    fprintf(out, "255\n");
    for (i = 1; i < n - 1; i++) {
        for (j = 1; j < n - 1; j++) {
            switch(grid[IDX(i, j, n)]) {
            case EMPTY:
                fprintf(out, "%c%c%c", 255, 255, 255);
                break;
            case TB:
                fprintf(out, "%c%c%c", 0, 0, 255);
                break;
            case LR:
                fprintf(out, "%c%c%c", 255, 0, 0);
                break;
            default:
                /*printf("Error: unknown cell state %u\nIndex: %d-%d\n", grid[IDX(i, j, n)], i - 1, j - 1);
                abort();*/
                fprintf(out, "%c%c%c", grid[IDX(i, j, n)], 0, 255 - grid[IDX(i, j, n)]);
            }
        }
    }
    fclose(out);
}

int main(int argc, char *argv[]) {
    cell_t *grid;

    cell_t *d_cur;
    cell_t *d_next;

    char buf[BUFLEN];
    int s;
    int N = 256;
    int nsteps = 512;
    float rho = 0.2;
    double tstart;
    double tend;

    if (argc > 4) {
        printf("Usage: %s [nsteps [rho [N]]]\n", argv[0]);
        return -1;
    }

    if (argc > 1) {
        nsteps = atoi(argv[1]);
    }

    if (argc > 2) {
        rho = atof(argv[2]);
    }

    if (argc > 3) {
        N = atoi(argv[3]);
    }

    /* |size| comprehends a layer of ghost cells. */
    const size_t size = (N + 2) * (N + 2) * sizeof(cell_t);

    /* Define block size and grid size for copying ghost cells on the sides of the grid. */
    dim3 cpy_block(BLKSIZE);
    dim3 cpy_grid((N + 2 + BLKSIZE - 1) / BLKSIZE);

    /* Define block size and grid size for calculating the steps. */
    dim3 step_block(BLKSIZE, BLKSIZE);
    dim3 step_grid((N + BLKSIZE - 1) / BLKSIZE, (N + BLKSIZE - 1) / BLKSIZE);

    /* Allocate grids. */
    grid = (cell_t*) malloc(size);

    hipMalloc((void **) &d_cur, size);
    hipMalloc((void **) &d_next, size);


    setup(grid, N + 2, rho);

    /* Dump the initialized grid. */
    snprintf(buf, BUFLEN, "cuda-traffic-start.ppm");
    dump(grid, N + 2, buf);

    /* Copy the initialized grid to the device. */
    hipMemcpy(d_cur, grid, size, hipMemcpyHostToDevice);

    tstart = hpc_gettime();
    for (s = 0; s < nsteps; s++) {
        /* Initialize the ghost cells in the first grid. */
        copy_top_bottom<<<cpy_grid, cpy_block>>>(d_cur, N);
        copy_left_right<<<cpy_grid, cpy_block>>>(d_cur, N);

        horizontal_step<<<step_grid, step_block>>>(d_cur, d_next, N);

    	  /* Initialize the ghost cells in the second grid. */
    	  copy_top_bottom<<<cpy_grid, cpy_block>>>(d_next, N);
    	  copy_left_right<<<cpy_grid, cpy_block>>>(d_next, N);

        vertical_step<<<step_grid, step_block>>>(d_next, d_cur, N);

        /* Dump each step. */
        /*hipMemcpy(grid, d_cur, size, hipMemcpyDeviceToHost);
        snprintf(buf, BUFLEN, "cuda-traffic-%05d.ppm", s);
        dump(grid, N + 2, buf);*/
    }
    hipDeviceSynchronize();
    tend = hpc_gettime();
    fprintf(stderr, "Execution time (s): %f\n", tend - tstart);

    /* Copy the result grid back to the host. */
    hipMemcpy(grid, d_cur, size, hipMemcpyDeviceToHost);

    /* Dump the last state. */
    snprintf(buf, BUFLEN, "cuda-traffic-%05d.ppm", s);
    dump(grid, N + 2, buf);

    /* Free memory. */
    free(grid);

    hipFree(d_cur);
    hipFree(d_next);

    return 0;
}
